#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdint>


struct Point {
    double x, y;
    uint32_t mass;
    double vx, vy;
};



__device__ double2 scalar_to_2d(double scl,
    const double2 start,
    const double2 target)
{
    double2 out_vec;
    double dx = target.x - start.x;
    double dy = target.y - start.y;
    double dist = sqrt(dx * dx + dy * dy);

    if (dist < 1e-9) {
        out_vec.x = 0.0;
        out_vec.y = 0.0;
        return out_vec;
    }

    double inv_dist = 1.0 / dist;
    out_vec.x = scl * dx * inv_dist;
    out_vec.y = scl * dy * inv_dist;
    return out_vec;
}



__device__ double compute_grav_force(Point p1, Point p2) {
    double dx = p1.x - p2.x;
    double dy = p1.y - p2.y;
    double distance_squared = dx * dx + dy * dy;
    if (distance_squared == 0) {
        return 0.0;
    }
    return p1.mass * p2.mass / distance_squared;
}

__global__ void compute_new(Point *points, uint8_t iterative_steps, double dt) {
    if ((blockIdx.y * blockDim.y + threadIdx.y) >= blockIdx.x * blockDim.x + threadIdx.x) {
        return;
    }
    
    Point &p1 = points[blockIdx.x * blockDim.x + threadIdx.x];
    Point &p2 = points[blockIdx.y * blockDim.y + threadIdx.y];
    
    double force = 0.0;
    force += compute_grav_force(p1, p2);
    double2 acc1 = scalar_to_2d(force / p1.mass, make_double2(p1.x, p1.y), make_double2(p2.x, p2.y));
    double2 acc2 = scalar_to_2d(force / p2.mass, make_double2(p2.x, p2.y), make_double2(p1.x, p1.y));
    atomicAdd(&p1.vx, acc1.x * dt);
    atomicAdd(&p1.vy, acc1.y * dt);
    atomicAdd(&p2.vx, acc2.x * dt);
    atomicAdd(&p2.vy, acc2.y * dt);
}

__global__ void _compute(Point *points, uint8_t iterative_steps, double dt, int n_points) {
    const constexpr uint64_t threads = 1024;
    const uint64_t blocks = (n_points + threads - 1) / threads;

    compute_new<<<blocks, threads>>>(points, iterative_steps, dt);
}

extern "C" void compute(Point* points, int n_points, uint8_t iterative_steps, double dt) {
    _compute<<<1, 1>>>(points, iterative_steps, dt, n_points);
    hipDeviceSynchronize();
}
